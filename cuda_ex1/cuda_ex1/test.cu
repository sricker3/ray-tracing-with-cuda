
#include "test.cuh"

CUDA_CALL Ball::Ball(int r)
{
	radius = r;
}

CUDA_CALL int Ball::getRadius()
{
	return radius;
}

CUDA_CALL void Ball::setRadius(int r)
{
	radius = r;
}


__device__ float getRad()
{
	return 1.0f/0.0f;
}