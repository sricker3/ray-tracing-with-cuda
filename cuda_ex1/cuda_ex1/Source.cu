#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_math_constants.h>
#include "test.cuh"

__global__ void add(int a, int b, int *c, Ball ba)
{
	*c = a + b;
	//*c += getRad();
	ba.setRadius(17);
	*c = ba.getRadius();
	int nein = 99;
	memcpy(c, &nein, sizeof(int));
}

__global__ void test_math(float a, float* ret)
{
	//*ret = sqrtf(a);
	//float inf = 1.0/0.0;
	float inf = HIP_INF;
	if(4 < inf)
		*ret = HIP_INF;
	else
		*ret = inf;
}

int main(int argc, char* agrv[])
{
	int host_c;
	int *dev_c;
	float* ret;
	float host_ret;
	Ball ball(5);
	ball.setRadius(3);

	hipMalloc((void**)&dev_c, sizeof(int));
	hipMalloc((void**)&ret, sizeof(float));
	add<<< 1,1 >>>(2, 7, dev_c, ball);
	hipMemcpy(&host_c, dev_c, sizeof(int), hipMemcpyDeviceToHost);
	test_math<<<1, 1>>>(7, ret);
	hipMemcpy(&host_ret, ret, sizeof(float), hipMemcpyDeviceToHost);

	std::cout << "2 + 7 + ball radius = " << host_c << std::endl;
	hipFree(dev_c);
	std::cout<<"sqrt(7) = "<<host_ret<<std::endl;
	printf("0x%08x\n", host_ret);

	hipFree(ret);

	std::cout << "Enter something then hit enter to close...\n";
	std::string input;
	std::cin >> input;

	return 0;
} //if rename a cpp file to a cu file, right click file to make sure it is being treated as a cu file!!!
//Note: if using avast, you should probably turn off deepscreen under settings->antivirus. Noticed it was causing my vs 2013 programs to run twice and they ran much slower. Still not sure why the debugger output says 2 threads exited.