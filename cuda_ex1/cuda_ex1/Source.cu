#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string>
#include <iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "test.cuh"

__global__ void add(int a, int b, int *c, Ball ba)
{
	*c = a + b;
	//*c += getRad();
	*c += ba.getRadius();
}

int main(int argc, char* agrv[])
{
	int host_c;
	int *dev_c;
	Ball ball(5);

	hipMalloc((void**)&dev_c, sizeof(int));
	add<<< 1,1 >>>(2, 7, dev_c, ball);
	hipMemcpy(&host_c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	std::cout << "2 + 7 + ball radius = " << host_c << std::endl;
	hipFree(dev_c);

	std::cout << "Enter something then hit enter to close...\n";
	std::string input;
	std::cin >> input;

	return 0;
} //if rename a cpp file to a cu file, right click file to make sure it is being treated as a cu file!!!
//Note: if using avast, you should probably turn off deepscreen under settings->antivirus. Noticed it was causing my vs 2013 programs to run twice and they ran much slower. Still not sure why the debugger output says 2 threads exited.