
#include <stdio.h>
#include <iostream>
#include <hip/hip_runtime.h>


__global__ void add(int a, int b, int *c)
{
	*c = a + b;
}

int main(int argc, char* agrv[])
{
	int host_c;
	int *dev_c;

	hipMalloc((void**)&dev_c, sizeof(int));
	add<<< 1,1 >>>(2, 7, dev_c);
	hipMemcpy(&host_c, dev_c, sizeof(int), hipMemcpyDeviceToHost);

	std::cout << "2 + 7 = " << host_c << std::endl;
	hipFree(dev_c);

	return 0;
} //if rename a cpp file to a cu file, right click file to make sure it is being treated as a cu file!!!