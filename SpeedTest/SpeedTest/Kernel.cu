#include "hip/hip_runtime.h"

#include "Kernel.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

__global__ void grayScaleConvertGPU(unsigned char* image, int dimx, int dimy, int d)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

    if(x < dimx && y < dimy)
    {
        unsigned int lum = image[d*(x + y*dimx) + 0]*.2126 + image[d*(x + y*dimx) + 1]*.7152 + image[d*(x + y*dimx) + 2]*.0722;
        image[d*(x + y*dimx) + 0] = (unsigned char) lum;
        image[d*(x + y*dimx) + 1] = (unsigned char) lum;
        image[d*(x + y*dimx) + 2] = (unsigned char) lum;
    }
}

void grayScaleConvert(unsigned char* image, int x, int y, int d)
{
    unsigned char* pixelBuffer;
    hipMalloc((void**)&pixelBuffer, sizeof(unsigned char)*x*y*d);
    hipMemcpy(pixelBuffer, image, sizeof(unsigned char)*x*y*d, hipMemcpyHostToDevice);

    dim3 blocks(roundf(x/16), roundf(y/16));
	dim3 threads(16, 16);
	grayScaleConvertGPU<<<blocks, threads>>>(pixelBuffer, x, y, d);

    hipMemcpy(image, pixelBuffer, sizeof(unsigned char)*x*y*d, hipMemcpyDeviceToHost);
	hipFree(pixelBuffer);
}